
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <time.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void random_ints(int* b, int size){
	for(int i = 0; i < size; i++){
		b[i] = rand();
	}
}

int main(){
	clock_t tStart = clock();
	int *a, *b, *c;
	int *_a, *_b, *_c;
	
	int size = N * sizeof(int);
	tStart = clock();
	hipMalloc((void **) &_a, size);
	hipMalloc((void **) &_b, size);
	hipMalloc((void **) &_c, size);
	
	printf("allocating mem device: %.2f\n", (double)(clock() - tStart));
	
	tStart = clock();
	
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size); 
	
        printf("allocating mem host: %.2f\n", (double)(clock() - tStart));
	
	tStart = clock();


	hipMemcpy(_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(_b, b, size, hipMemcpyHostToDevice);

	printf("allocating mem to host: %.2f\n", (double)(clock() - tStart));
	tStart = clock();


	add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(_a, _b, _c);
        printf("execution: %.2f\n", (double)(clock() - tStart));

	tStart = clock();
	

	hipMemcpy(c, _c, size, hipMemcpyDeviceToHost);
        printf("allocating mem back to host: %.2f\n", (double)(clock() - tStart));

	
	free(a);
	free(b);
	free(c);
	hipFree(_a);
	hipFree(_b);
	hipFree(_c);
	
	return 0;


}
